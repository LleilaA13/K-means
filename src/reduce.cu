#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <ctype.h>
#include <math.h>
#include <time.h>
#include <string.h>
#include <float.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>


#define CHECK_CUDA_CALL(a)                                                                            \
	{                                                                                                 \
		hipError_t ok = a;                                                                           \
		if (ok != hipSuccess)                                                                        \
			fprintf(stderr, "-- Error CUDA call in line %d: %s\n", __LINE__, hipGetErrorString(ok)); \
	}


__device__ __forceinline__ float warp_reduce(float val)
{
	FULL_MASK = 0xffffffff;
# pragma unroll
	for (unsigned int i = 16; i > 0; i /= 2)
	{
		val = max(val, __shfl_down_sync(FULL_MASK, val, i));
	}
	return val;
}

__global__ void reduce(float* inputs, unsigned int input_size, float* outputs)
{
	/* Eccoci qui all'interno della reduce più veloce del west. Questa implementazione è presa da questo blog:
	 * https://ashvardanian.com/posts/cuda-parallel-reductions/
	 * Praticamente questa implementazione sfrutta delle operazioni che vengono eseguite a livello dei warp. Se vi ricordate, in cuda
	 * i warp sono il più basso livello logico in cui le istruzioni vengono eseguite.
	 * ATTENZIONE: per fare si che questo algoritmo funzioni, input_size DEVE essere una potenza di 2, quindi dovete paddare il vostro array finché non ha
	 * la grandezza desiderata. Questo non influisce sulla correttezza del vostro algoritmo, vi dovete solo ricordare di paddare con un valore neutro per
	 * la vostra operazione (nel caso del MAX il valore è -FLT_MAX oppure semplicemente FLT_MIN)
	 */
    float sum = FLT_MIN;
    for (unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
            i < input_size;
            i += blockDim.x * gridDim.x)
        sum = max(sum, inputs[i]); // Questo for serve in caso non abbiate abbastanza thread per parallelizzare, e quindi ogni thread deve gestire più elementi. Per fortuna non è il vostro caso, quindi questo for in realtà di riduce semplicemente a sum += inputs[i] (fate la prova togliendolo per vedere che effettivamente l'algoritmo funziona lo stesso)

    __shared__ float shared[32];
    unsigned int lane = threadIdx.x % warpSize;
    unsigned int wid = threadIdx.x / warpSize;

    sum = warp_reduce(sum);
    if (lane == 0)
        shared[wid] = sum;

    // Wait for all partial reductions
    __syncthreads();

    sum = (threadIdx.x < blockDim.x / warpSize) ? shared[lane] : 0;
    if (wid == 0)
        sum = warp_reduce(sum);

    if (threadIdx.x == 0)
        outputs[blockIdx.x] = sum;
}


int main(int argc, char** argv)
{
  float h_array = {1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0,
                   1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0,
                   1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0,
                   1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0,
                   1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0,
                   1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0,
                   1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0,
                   1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0};
  float* d_array;
  float* d_res;
  float res = 0.0f;
  float*
  for (unsigned int i = 0; i < 64; ++i)
  {
    res += h_array[i];
  }
  std::cout << "cpu result: " << res << std::endl;

  CHECK_CUDA_CALL(hipMalloc(&d_array, 64 * sizeof(float)))
  CHECK_CUDA_CALL(hipMalloc(&d_res, sizeof(float)))
  CHECK_CUDA_CALL(hipMemcpy(d_array, &array, 64 * sizeof(float), hipMemcpyHostToDevice))
  CHECK_CUDA_CALL(hipMemset(d_res, 0.0f, sizeof(float)))

  reduce<<<1, 64>>>(d_array, 64, d_res);
  /* ATTENZIONE: Nel caso vi trovaste nella necessità di lanciare più di un blocco, vi dovete ricordare di effettuare l'operazione DUE VOLTE,
   * perché il primo risultato è una riduzione parziale. Quindi, se lanciate una griglia di 4 blocchi, d_res deve essere un array di 4 elementi
   * e la chiamata sarà una roba del genere:
   * reduce<<<4, 64>>>(d_array, 64, d_res);
   * reduce<<<1, 4>>>(d_res, 4, d_res);
   * fate attenzione che la seconda chiamata è un solo blocco di grandezza dimGrid (in questo caso 4)
   */
  CHECK_CUDA_CALL(hipDeviceSynchronize())

  float* h_res;

  CHECK_CUDA_CALL(hipMemcpy(h_res, d_res, sizeof(float), cudaMemcyDeviceToHost))

  std::cout << "gpu result: " << h_res << std::endl;

  return EXIT_SUCCESS;
}
